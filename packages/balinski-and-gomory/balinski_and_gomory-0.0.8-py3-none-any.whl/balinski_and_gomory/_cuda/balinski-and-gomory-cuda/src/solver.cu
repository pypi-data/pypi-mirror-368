#include "hip/hip_runtime.h"
#include <iostream>
// #include <cstdlib>
// #include <ctime>
#include <hip/hip_runtime.h>
#include <float.h>
#include <hipcub/hipcub.hpp>


// Forward declaration
// void solve(float* d_C, int* d_X, float* d_U, float* d_V, int n);
// #include <hip/hip_runtime.h>
// #include <>
// #include <iostream>

#define IDX2C(i,j,n) ((j)*(n)+(i))
// #define IDX2C(i,j,n) ((i)*(n)+(j))

// Device kernels
__global__ void compute_B(const float* C, const float* U, const float* V, float* B, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        B[IDX2C(i, j, n)] = C[IDX2C(i, j, n)] - U[i] - V[j];
    }
}

__global__ void find_argmin(const float* B, int* out_idx, float* out_val, int n) {
    __shared__ float min_val[256];
    __shared__ int min_idx[256];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int total = n * n;

    float val = (index < total) ? B[index] : FLT_MAX;
    min_val[tid] = val;
    min_idx[tid] = index;
    __syncthreads();

    // Parallel reduction to find min
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && min_val[tid + s] < min_val[tid]) {
            min_val[tid] = min_val[tid + s];
            min_idx[tid] = min_idx[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out_val[blockIdx.x] = min_val[0];
        out_idx[blockIdx.x] = min_idx[0];
    }
}


__global__ void compute_col_to_row(int n, const int* X, int* col_to_row) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= n) return;

    for (int i = 0; i < n; ++i) {
        if (X[IDX2C(i, j, n)] == 1) {
            col_to_row[j] = i;
            return;
        }
    }
}

__global__ void solve_1bc_kernel(
    int n,
    const int* col_to_row,
    int k,
    const float* B,
    int* R,
    int* Q,
    bool* changed
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // rows
    int j = blockIdx.x * blockDim.x + threadIdx.x; // columns

    if (i >= n || j >= n) return;

    // Step (b): still only run one thread per column
    if (i == 0 && Q[j] != n) {
        int row = col_to_row[j];
        if (R[row] == n) {
            R[row] = j;
            *changed = true;
        }
    }

    // Step (c): one thread per (i, j)
    if (i != k && R[i] != n && Q[j] == n) {
        float b_val = B[IDX2C(i, j, n)];
        if (b_val == 0.0f) {
            if (atomicMin(&Q[j], i) > i) {
                *changed = true;
            }
            // if (atomicCAS(&Q[j], n, i) == n) {
            //     *changed = true;
            // }
            // if (Q[j] == n) {
            //     Q[j] = i;
            //     *changed = true;
            // }
        }
    }
}


void solve_1bc(
    int n,
    int* d_col_to_row,
    int* k,            // now pointer
    int* l,            // now pointer (not used here, but kept for symmetry)
    float* d_B,
    int* d_R,
    int* d_Q
){
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + 15) / 16, (n + 15) / 16);

    bool h_changed;
    bool* d_changed;
    hipMalloc(&d_changed, sizeof(bool));

    do {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);

        // NOTE: kernel takes k by value — pass *k
        solve_1bc_kernel<<<numBlocks, threadsPerBlock>>>(
            n, d_col_to_row, *k, d_B, d_R, d_Q, d_changed
        );

        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    } while (h_changed);

    hipFree(d_changed);
}



__global__ void set_array_value(int* arr, int value, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) arr[idx] = value;
}

__global__ void update_duals(int* R, int* Q, float* U, float* V, float epsilon, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (R[i] != n) U[i] += epsilon;
        if (Q[i] != n) V[i] -= epsilon;
    }
}

bool solve_from_kl(
    int n,
    float* d_C, int* d_X, int* k, int* l, // now k and l are pointers
    float* d_U, float* d_V, float* d_B
) {
    // Allocate and initialize R and Q
    int* d_R; int* d_Q;
    hipMalloc(&d_R, n * sizeof(int));
    hipMalloc(&d_Q, n * sizeof(int));

    set_array_value<<<(n + 255)/256, 256>>>(d_R, n, n);
    set_array_value<<<(n + 255)/256, 256>>>(d_Q, n, n);

    // Q[*l] = *k
    hipMemcpy(&d_Q[*l], k, sizeof(int), hipMemcpyHostToDevice);

    // Step 1: Solve 1BC
    int* d_col_to_row;
    hipMalloc(&d_col_to_row, n * sizeof(int));
    compute_col_to_row<<<(n + 255) / 256, 256>>>(n, d_X, d_col_to_row);
    hipDeviceSynchronize();

    for (int s = 0; s < n; ++s) {
        solve_1bc(n, d_col_to_row, k, l, d_B, d_R, d_Q);
    }
    hipFree(d_col_to_row);

    // Step 2: Check if R[*k] != n and R[*k] != *l
    int h_Rk;
    hipMemcpy(&h_Rk, &d_R[*k], sizeof(int), hipMemcpyDeviceToHost);

    if (h_Rk != n && h_Rk != *l) {
        int k_ = *k;
        int l_ = *l;

        int h_R, h_Q;

        while (true) {
            // X[k_, l_] = 1
            int one = 1;
            int idx_on = IDX2C(k_, l_, n);
            hipMemcpy(&d_X[idx_on], &one, sizeof(int), hipMemcpyHostToDevice);

            // l_ = R[k_]
            hipMemcpy(&h_R, &d_R[k_], sizeof(int), hipMemcpyDeviceToHost);
            l_ = h_R;

            // X[k_, l_] = 0
            int zero = 0;
            int idx_off = IDX2C(k_, l_, n);
            hipMemcpy(&d_X[idx_off], &zero, sizeof(int), hipMemcpyHostToDevice);

            // k_ = Q[l_]
            hipMemcpy(&h_Q, &d_Q[l_], sizeof(int), hipMemcpyDeviceToHost);
            k_ = h_Q;

            if (k_ == *k && l_ == *l)
                break;
        }

        float b_kl;
        hipMemcpy(&b_kl, &d_B[IDX2C(*k, *l, n)], sizeof(float), hipMemcpyDeviceToHost);
        float epsilon = -b_kl;

        float v_l;
        hipMemcpy(&v_l, &d_V[*l], sizeof(float), hipMemcpyDeviceToHost);
        v_l -= epsilon;
        hipMemcpy(&d_V[*l], &v_l, sizeof(float), hipMemcpyHostToDevice);

        // Recompute B = C - U.unsqueeze(1) - V
        dim3 threads(16, 16);
        dim3 blocks((n + 15) / 16, (n + 15) / 16);
        compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);

        *k = n;
        *l = n;

        // float* h_B = new float[n * n];
        // hipMemcpy(h_B, d_B, sizeof(float) * n * n, hipMemcpyDeviceToHost);
        // int min_idx = 0;
        // float min_val = h_B[0];
        // for (int idx = 1; idx < n * n; ++idx) {
        //     if (h_B[idx] < min_val) {
        //         min_val = h_B[idx];
        //         min_idx = idx;
        //     }
        // }
        // delete[] h_B;
        // // Update k, l in-place
        // *k = min_idx % n;
        // *l = min_idx / n;

        hipFree(d_R);
        hipFree(d_Q);
        return true;
    }

    // Branch B: find epsilon satisfying mask
    float epsilon = -1;
    bool found = false;

    for (int i = 0; i < n; ++i) {
        int r_val;
        hipMemcpy(&r_val, &d_R[i], sizeof(int), hipMemcpyDeviceToHost);
        if (r_val == n) continue;

        for (int j = 0; j < n; ++j) {
            int q_val;
            hipMemcpy(&q_val, &d_Q[j], sizeof(int), hipMemcpyDeviceToHost);
            if (q_val != n) continue;

            float b_ij;
            hipMemcpy(&b_ij, &d_B[IDX2C(i, j, n)], sizeof(float), hipMemcpyDeviceToHost);
            if (b_ij >= 0 && (!found || b_ij < epsilon)) {
                epsilon = b_ij;
                found = true;
            }
        }
    }

    if (!found) {
        float b_kl;
        hipMemcpy(&b_kl, &d_B[IDX2C(*k, *l, n)], sizeof(float), hipMemcpyDeviceToHost);
        epsilon = -b_kl;
    }

    // Update duals
    update_duals<<<(n + 255) / 256, 256>>>(d_R, d_Q, d_U, d_V, epsilon, n);

    // Recompute B
    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);
    compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);
    hipDeviceSynchronize();

    // Check B[*k,*l]
    float b_kl_check;
    hipMemcpy(&b_kl_check, &d_B[IDX2C(*k, *l, n)], sizeof(float), hipMemcpyDeviceToHost);

    if (b_kl_check < 0) {
        hipFree(d_R);
        hipFree(d_Q);
        return true;
    }

    // Check if any B[i,j] < 0
    bool any_negative = false;
    for (int i = 0; i < n && !any_negative; ++i) {
        for (int j = 0; j < n; ++j) {
            float b_ij;
            hipMemcpy(&b_ij, &d_B[IDX2C(i, j, n)], sizeof(float), hipMemcpyDeviceToHost);
            if (b_ij < 0) {
                any_negative = true;
                hipFree(d_R);
                hipFree(d_Q);
                *k = i;
                *l = j;
                return true;
            }
        }
    }

    hipFree(d_R);
    hipFree(d_Q);
    return false;
}

// using KVP = hipcub::KeyValuePair<int, float>;


void solve(float* d_C, int* d_X, float* d_U, float* d_V, int n) {
    size_t sizeMat = n * n * sizeof(float);

    // Allocate B
    float* d_B;
    hipMalloc(&d_B, sizeMat);

    // Allocate buffers for argmin
    int* d_idx; float* d_val;
    hipMalloc(&d_idx, sizeof(int));
    hipMalloc(&d_val, sizeof(float));

    // Compute initial B
    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);
    compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);

    const size_t num_items = static_cast<size_t>(n) * static_cast<size_t>(n);
    void* d_temp = nullptr; size_t temp_bytes = 0;
    hipcub::DeviceReduce::ArgMin(nullptr, temp_bytes, d_B, d_val, d_idx, num_items);
    hipMalloc(&d_temp, temp_bytes);

    int steps = 0;
    int k = n;
    int l = n;
    while (true) {
        if (k == n){
            // const size_t num_items = static_cast<size_t>(n) * static_cast<size_t>(n);
            // void* d_temp = nullptr; size_t temp_bytes = 0;
            // hipcub::DeviceReduce::ArgMin(nullptr, temp_bytes, d_B, d_val, d_idx, num_items);
            // hipMalloc(&d_temp, temp_bytes);
            hipcub::DeviceReduce::ArgMin(d_temp, temp_bytes, d_B, d_val, d_idx, num_items);
            // hipFree(d_temp);

            // int   h_idx;
            // hipMemcpy(&h_idx, d_idx, sizeof(int), hipMemcpyDeviceToHost);

            // int k = h_idx / n;
            // int l = h_idx % n;
            // Find argmin(B)
            // int totalThreads = n * n;
            // int blockSize = 256;
            // int gridSize = (totalThreads + blockSize - 1) / blockSize;
            // find_argmin<<<gridSize, blockSize>>>(d_B, d_idx, d_val, n);

            int h_idx;
            // float h_val;
            hipMemcpy(&h_idx, d_idx, sizeof(int), hipMemcpyDeviceToHost);
            // hipMemcpy(&h_val, d_val, sizeof(float), hipMemcpyDeviceToHost);

            // k = h_idx / n;
            // l = h_idx % n;
            k = h_idx % n;
            l = h_idx / n;
        }


        // Call solve_from_kl, which returns false if we should stop
        bool should_continue = solve_from_kl(n, d_C, d_X, &k, &l, d_U, d_V, d_B);
        steps++;
        std::cout << "Step " << steps << ": argmin at B[" << k << "][" << l << "] \n";

        if (!should_continue) {
            std::cout << "Solver has converged after " << steps << " steps.\n";
            break;
        }
    }

    // Cleanup
    hipFree(d_B);
    hipFree(d_temp);
    // hipFree(d_out);
    hipFree(d_idx);
    hipFree(d_val);
}


__global__ void check_feasible_condition(const float* C, const int* X, const float* U, const float* V, int* out, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        if (X[IDX2C(i, j, n)] == 1) {
            float diff = C[IDX2C(i, j, n)] - U[i] - V[j];
            if (fabsf(diff) > 1e-4f) {
                atomicExch(out, 1); // feasible violated
            }
        }
    }
}

__global__ void check_slack_condition(const float* C, const float* U, const float* V, int* out, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        float diff = C[IDX2C(i, j, n)] - U[i] - V[j];
        if (diff < -1e-4f) {
            atomicExch(out, 1); // slack violated
        }
    }
}

void verify_solution(float* d_C, int* d_X, float* d_U, float* d_V, int n) {
    int h_feasible = 0, h_slack = 0;
    int *d_feasible, *d_slack;
    hipMalloc(&d_feasible, sizeof(int));
    hipMalloc(&d_slack, sizeof(int));
    hipMemset(d_feasible, 0, sizeof(int));
    hipMemset(d_slack, 0, sizeof(int));

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);

    check_feasible_condition<<<blocks, threads>>>(d_C, d_X, d_U, d_V, d_feasible, n);
    check_slack_condition<<<blocks, threads>>>(d_C, d_U, d_V, d_slack, n);

    hipMemcpy(&h_feasible, d_feasible, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_slack, d_slack, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_feasible);
    hipFree(d_slack);

    bool feasible_ok = (h_feasible == 0);
    bool slack_ok = (h_slack == 0);

    std::cout << "\n=== Post-Solution Verification ===\n";
    std::cout << "feasible condition: " << (feasible_ok ? "PASS" : "FAIL") << "\n";
    std::cout << "slack condition: " << (slack_ok ? "PASS" : "FAIL") << "\n";
    std::cout << "Overall check: " << ((feasible_ok && slack_ok) ? "✓ Passed" : "✗ Failed") << "\n\n";
}