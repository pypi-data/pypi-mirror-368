#include <iostream>
#include <iostream>
#include <cstdlib>
#include <ctime>

#include "solver.h"

#define IDX2C(i,j,n) ((j)*(n)+(i))

// extern void solve(const float*, const float*, float*, int, int, int);
// extern void solve(float* d_C, int* d_X, float* d_U, float* d_V, int n);


void fill_random(float* matrix, int n) {
    for (int i = 0; i < n * n; ++i)
        matrix[i] = static_cast<float>(rand() % 100); // 0–9
}

void initialize_identity_mask(int* X, int n) {
    for (int i = 0; i < n * n; ++i)
        X[i] = 0;
    for (int i = 0; i < n; ++i)
        X[IDX2C(i, i, n)] = 1;
}

void compute_V_from_C_and_X(const float* C, const int* X, float* V, int n) {
    for (int j = 0; j < n; ++j) {
        float sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += C[IDX2C(i, j, n)] * X[IDX2C(i, j, n)];
        }
        V[j] = sum;
    }
}

template <typename T>
void print_matrix(const T* matrix, int n, const char* name) {
    std::cout << name << " (" << n << "x" << n << "):\n";
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            std::cout << matrix[IDX2C(i, j, n)] << " ";
        std::cout << "\n";
    }
    std::cout << std::endl;
}


void print_vector(const float* vec, int n, const char* name) {
    std::cout << name << " (" << n << "): ";
    for (int i = 0; i < n; ++i)
        std::cout << vec[i] << " ";
    std::cout << "\n\n";
}

int main(int argc, char* argv[]) {
    int n = 100; // default

    if (argc >= 2) {
        if (std::string(argv[1]) == "-n") {
            if (argc >= 3) {
                n = std::atoi(argv[2]);
            } else {
                std::cerr << "Error: -n requires a value.\n";
                return 1;
            }
        } else {
            n = std::atoi(argv[1]); // simple style
        }
    }

    // std::cout << "Matrix size n = " << n << std::endl;
    std::cout << "Matrix size n = " << n << std::endl;

    size_t matSize = n * n * sizeof(float);
    size_t maskSize = n * n * sizeof(int);
    size_t vecSize = n * sizeof(float);

    // Host allocations
    float* h_C = new float[n * n];
    int* h_X = new int[n * n];
    float* h_U = new float[n];
    float* h_V = new float[n];

    // Fill values
    srand(static_cast<unsigned>(time(0)));
    fill_random(h_C, n);
    initialize_identity_mask(h_X, n);

    for (int i = 0; i < n; ++i) h_U[i] = 0; // U = 0
    compute_V_from_C_and_X(h_C, h_X, h_V, n);

    // Print initialized values
    print_matrix(h_C, n, "Matrix C");
    // print_matrix(reinterpret_cast<float*>(h_X), n, "Mask X");
    print_matrix(h_X, n, "Mask X");
    print_vector(h_U, n, "Vector U");
    print_vector(h_V, n, "Vector V");

    // Device allocations
    float *d_C, *d_U, *d_V;
    int* d_X;
    hipMalloc(&d_C, matSize);
    hipMalloc(&d_X, maskSize);
    hipMalloc(&d_U, vecSize);
    hipMalloc(&d_V, vecSize);

    // Copy to device
    hipMemcpy(d_C, h_C, matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_X, h_X, maskSize, hipMemcpyHostToDevice);
    hipMemcpy(d_U, h_U, vecSize, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, vecSize, hipMemcpyHostToDevice);

    // Call solver
    // solve(d_C, d_X, d_U, d_V, n);
    // verify_solution(d_C, d_X, d_U, d_V, n);

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsed_ms = 0.0f;

    // ---- Time solve() ----
    hipEventRecord(start);
    solve(d_C, d_X, d_U, d_V, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_ms, start, stop);
    std::cout << "solve() time: " << elapsed_ms << " ms" << std::endl;

    // ---- Time verify_solution() ----
    hipEventRecord(start);
    verify_solution(d_C, d_X, d_U, d_V, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_ms, start, stop);
    std::cout << "verify_solution() time: " << elapsed_ms << " ms" << std::endl;

    // Destroy events after use
    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(h_X, d_X, maskSize, hipMemcpyDeviceToHost);
    // print_matrix(reinterpret_cast<float*>(h_X), n, "Mask X");
    // print_matrix(h_X, n, "Mask X");
    // print_vector(h_U, n, "Vector U");
    // print_vector(h_V, n, "Vector V");


    // Cleanup
    hipFree(d_C);
    hipFree(d_X);
    hipFree(d_U);
    hipFree(d_V);
    delete[] h_C;
    delete[] h_X;
    delete[] h_U;
    delete[] h_V;

    return 0;
}
